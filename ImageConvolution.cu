#include "hip/hip_runtime.h"
//AUTHOR : HAMORA HADI

#include<stdio.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<assert.h>

#define Mask_width  64
#define Mask_height 64
#define Mask_radius_x Mask_width/2
#define Mask_radius_y Mask_height/2
#define TILE_WIDTH 32  
#define w_x (TILE_WIDTH + Mask_width - 1) 
#define w_y (TILE_WIDTH + Mask_height - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

float convolution_2D_OnHost(float * N,float * M,int width, int height,int i,int j);

#define wbCheck(stmt)  do {                                                    
        hipError_t err = stmt;                                               
        if (err != hipSuccess) {                                             
            printf( "Failed to run stmt %d ", __LINE__);                       
            printf( "Got CUDA error ...  %s ", hipGetErrorString(err));    
            return -1;                                                        
        }                                                                     
    } while(0)

__global__ void convolution(float *I, const float* __restrict__ M, float *P,int channels, int width, int height){
   __shared__ float N_ds[w_y][w_x];
   int k;
   for (k = 0; k < channels; k++){
      int dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
         destY = dest / w_x, destX = dest % w_x,
         srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius_x,
         srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius_y,
         src = (srcY * width + srcX) * channels + k;
      if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
         N_ds[destY][destX] = I[src];
      else
         N_ds[destY][destX] = 0.0;

        for (int iter=1; iter <= (w_x * w_y) / (TILE_WIDTH*TILE_WIDTH); iter++){
          // Second batch loading
          dest = threadIdx.y * TILE_WIDTH + threadIdx.x + iter*(TILE_WIDTH * TILE_WIDTH);
            destY = dest / w_x, destX = dest % w_x;
            srcY  = blockIdx.y * TILE_WIDTH + destY - Mask_radius_x;
            srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius_y;
            src = (srcY * width + srcX) * channels + k;
            if (destY < w_y && destX < w_x)
            {
                if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                    N_ds[destY][destX] = I[src];
                else
                    N_ds[destY][destX] = 0.0;
            }
        }
      __syncthreads();

      float accum = 0;
      int y, x;
      for (y = 0; y < Mask_width; y++)
         for (x = 0; x < Mask_width; x++)
            accum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * Mask_width + x];
      y = blockIdx.y * TILE_WIDTH + threadIdx.y;
      x = blockIdx.x * TILE_WIDTH + threadIdx.x;
      if (y < height && x < width)
         //P[(y * width + x) * channels + k] = clamp(accum);
    	  P[(y * width + x) * channels + k] = accum;
      __syncthreads();
   }
}

float convolution_2D_OnHost(float * N,float * M,int width, int height,int i,int j);

int main() {
    int maskRows=Mask_height;
    int maskColumns=Mask_width;

    int imageChannels=1;
    int imageWidth=2048;
    int imageHeight=2048;

    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;
    float * outputImageOnHost;

    hostMaskData = (float *) malloc(sizeof(float)*maskRows*maskColumns);
    for(int i=0;i<maskRows*maskColumns;i++)
    	hostMaskData[i]=1.0;
    }
    //assert(maskRows == 5); 
    //assert(maskColumns == 5); 

    //To store Memory
    hostInputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight);
    hostOutputImageData = (float *) malloc(sizeof(float)*imageWidth*imageHeight);
    outputImageOnHost = (float *) malloc(sizeof(float)*imageWidth*imageHeight);
    for(int i=0;i<imageWidth*imageHeight;i++)//To set Image data as 1.0{
    	hostInputImageData[i]=1.0;
    }

    wbCheck(hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));
    wbCheck(hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float)));
    wbCheck(hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float)));

    wbCheck(hipMemcpy(deviceInputImageData,
               hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(deviceMaskData,
               hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice));

    dim3 dimGrid(((imageWidth-1)/TILE_WIDTH)+1, ((imageHeight-1)/TILE_WIDTH)+1,1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    convolution<<<dimGrid,dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData,
                                       imageChannels, imageWidth, imageHeight);

	hipError_t err1 = hipPeekAtLastError();
	hipDeviceSynchronize();
	printf( "Got CUDA error ... %s \n", hipGetErrorString(err1));

    hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);

#if 0  //
    for(int i=0;i<imageWidth*imageHeight;i++){
        if(i>0 && (i%imageWidth==0))
            printf("\n");
    		printf("%0.2f \t",*(hostOutputImageData+i));

    		}
#endif
    for(int i=0;i<imageWidth;i++){
        			for(int j=0;j<imageHeight;j++){
        				outputImageOnHost[(i*imageWidth)+j]=convolution_2D_OnHost(hostInputImageData,hostMaskData,imageWidth,imageHeight,i,j);
        			}
        		}

#if 0  //
    for(int i=0;i<imageWidth*imageHeight;i++){
    		if(i>0 && (i%imageWidth==0))
    			printf("\n");
    		printf("%0.2f \t",*(outputImageOnHost+i));

    		}
#endif


        for(int i=0;i<imageWidth*imageHeight;i++){
        	if(outputImageOnHost[i]!=hostOutputImageData[i]){
        		 printf("\nMismatch at (Row,Col) = [%d][%d], hostComputed[]: %0.0f And device[]: %0.0f", i / imageWidth, i % imageHeight, outputImageOnHost[i], hostOutputImageData[i]);
            }
        }

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    free(hostInputImageData);
    free(hostOutputImageData);

    printf("\n Number of Threads Per Block created in code: %d",TILE_WIDTH*TILE_WIDTH);
    printf("\n Number of Blocks Created :%d",(((imageWidth-1)/TILE_WIDTH)+1)*(((imageWidth-1)/TILE_WIDTH)+1));
    printf("No Error");
    return 0;
}

float convolution_2D_OnHost(float * N,float * M,int width, int height,int i,int j){
	float Pvalue=0.0;
	int N_start_point_i = i - (Mask_width/2);
	int N_start_point_j = j - (Mask_height/2);

	for(int k=0;k<Mask_width;k++){
		for(int l=0;l<Mask_height;l++){
			if(((N_start_point_i+k)>=0) && ((N_start_point_i+k)<width)&&((N_start_point_j+l)>=0)&&((N_start_point_j+l)<height)){
			    Pvalue+=N[(N_start_point_i+k)*width+(N_start_point_j+l)]*M[(k*Mask_width)+l];
			}
		}
	}

	return((Pvalue));
}
